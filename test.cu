#include "hip/hip_runtime.h"
__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index/spatial)%filters;

    x[index] = (x[index] - mean[f])/(sqrtf(variance[f] + .00001f));
}
__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    if(offset < size) output[(batch*n+filter)*size + offset] *= biases[filter];
}
__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= n*size*batch) return;
    int i = index % size;
    index /= size;
    int j = index % n;
    index /= n;
    int k = index;

    output[(k*n+j)*size + i] += biases[j];
}

extern "C" void normalize_gpu(float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    size_t N = batch*filters*spatial;
    //printf("%d ", N);
    normalize_kernel<<<cuda_gridsize(N), BLOCK>>>(N, x, mean, variance, batch, filters, spatial);
    check_error(hipPeekAtLastError());
}
void scale_bias_gpu(float *output, float *biases, int batch, int n, int size)
{
    dim3 dimGrid((size-1)/BLOCK + 1, n, batch);
    dim3 dimBlock(BLOCK, 1, 1);
    //printf("%d ", batch*n*size);
    scale_bias_kernel<<<dimGrid, dimBlock>>>(output, biases, n, size);
    check_error(hipPeekAtLastError());
}
void add_bias_gpu(float *output, float *biases, int batch, int n, int size)
{
    int num = n*size*batch;
    //printf("%d\n", num);
    add_bias_kernel<<<cuda_gridsize(num), BLOCK>>>(output, biases, batch, n, size);
    check_error(hipPeekAtLastError());
}
